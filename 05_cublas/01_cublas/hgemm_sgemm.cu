#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#define M 3
#define K 4
#define N 2

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

#define CHECK_CUBLAS(call) { \
    hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        fprintf(stderr, "cuBLAS error in %s:%d: %d\n", __FILE__, __LINE__, status); \
        exit(EXIT_FAILURE); \
    } \
}


#define PRINT_MATRIX(mat, rows, cols) \
    for (int i = 0; i < rows; i++) { \
        for (int j = 0; j < cols; j++) \
            printf("%8.3f ", mat[i * cols + j]); \
        printf("\n"); \
    } \
    printf("\n");

void cpu_matmul(float *A, float *B, float *C) {
    for (int i = 0; i < M; i++)
        for (int j = 0; j < N; j++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++)
                sum += A[i * K + k] * B[k * N + j];
            C[i * N + j] = sum;
        }
}


int main(){
    float A[M * K] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f, 10.0f, 11.0f, 12.0f};
    float B[K * N] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f};
    float C_cpu[M*N] , C_cublas_sgemm[M*N] , C_cublas_hgemm[M*N];

    cpu_matmul(A, B, C_cpu);
    

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    float *d_A,*d_B,*d_C;
    CHECK_CUDA(hipMalloc(&d_A,sizeof(float)*M*K));
    CHECK_CUDA(hipMalloc(&d_B,sizeof(float)*K*N));
    CHECK_CUDA(hipMalloc(&d_C,sizeof(float)*M*N));

    CHECK_CUDA(hipMemcpy(d_A,&A,sizeof(float)*M*K,hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B,&B,sizeof(float)*K*N,hipMemcpyHostToDevice));

    float alpha=1.0f , beta=0.0f;
    CHECK_CUBLAS(hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,M,N,K,&alpha,d_A,M,d_B,K,&beta,d_C,M));
    CHECK_CUDA(hipMemcpy(C_cublas_sgemm,d_C,sizeof(float)*M*N,hipMemcpyDeviceToHost));


    //Hgemm
    half C_h[M*N];
    half *d_A_h,*d_B_h,*d_C_h;
    CHECK_CUDA(hipMalloc(&d_A_h,sizeof(half)*M*K));
    CHECK_CUDA(hipMalloc(&d_B_h,sizeof(half)*K*N));
    CHECK_CUDA(hipMalloc(&d_C_h,sizeof(half)*M*N));


    // Convert to half precision on CPU
    half A_h[M * K], B_h[K * N];
    for (int i = 0; i < M * K; i++) {
        A_h[i] = __float2half(A[i]);
    }
    for (int i = 0; i < K * N; i++) {
        B_h[i] = __float2half(B[i]);
    }

    CHECK_CUDA(hipMemcpy(d_A_h,&A_h,sizeof(half)*M*K,hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B_h,&B_h,sizeof(half)*K*N,hipMemcpyHostToDevice));

    half alpha_h=__float2half(1.0f) , beta_h=__float2half(0.0f);
    CHECK_CUBLAS(hipblasHgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,M,N,K,&alpha_h,d_A_h,M,d_B_h,K,&beta_h,d_C_h,M));
    CHECK_CUDA(hipMemcpy(C_h,d_C_h,sizeof(half)*M*N,hipMemcpyDeviceToHost));

    // Convert to float on CPU
    for (int i = 0; i < M * N; i++) {
        C_cublas_hgemm[i] = __half2float(C_h[i]);
    }


    // Print results
    printf("Matrix A (%dx%d):\n", M, K);
    PRINT_MATRIX(A, M, K);
    printf("Matrix B (%dx%d):\n", K, N);
    PRINT_MATRIX(B, K, N);
    printf("CPU Result (%dx%d):\n", M, N);
    PRINT_MATRIX(C_cpu, M, N);
    printf("cuBLAS SGEMM Result (%dx%d):\n", M, N);
    PRINT_MATRIX(C_cublas_sgemm, M, N);
    printf("cuBLAS HGEMM Result (%dx%d):\n", M, N);
    PRINT_MATRIX(C_cublas_hgemm, M, N);

    // Clean up
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipFree(d_A_h));
    CHECK_CUDA(hipFree(d_B_h));
    CHECK_CUDA(hipFree(d_C_h));
    CHECK_CUBLAS(hipblasDestroy(handle));

    return 0;

}