#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)

template <typename T>
void check(T err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line, static_cast<unsigned int>(err), hipGetErrorString(err), func);
        exit(EXIT_FAILURE);
    }
}


__global__ void  kernel1(float *A,int numOfElements){
    int idx= blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<numOfElements){
        A[idx]=A[idx]+1.0f;
    }
}

__global__ void kernel2(float *A,int numOfElements){
    int idx= blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<numOfElements){
        A[idx]=A[idx]+1.0f;
    }
}


void CUDART_CB myStreamCallback(hipStream_t stream, hipError_t status, void *userData) {
    printf("Stream callback: Operation completed\n");
}


int main(){
    int numOfElements=1000000;
    float *h_A, *h_B;
    float *d_A,*d_B;
    size_t size =numOfElements * sizeof(float);
    hipStream_t stream1,stream2;
    
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_A, size));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_B, size));

    CHECK_CUDA_ERROR(hipHostMalloc((void **)&h_A, size, hipHostMallocDefault));
    CHECK_CUDA_ERROR(hipHostMalloc((void **)&h_B, size, hipHostMallocDefault));

    int leastpriority,greatestpriority;
    CHECK_CUDA_ERROR(hipDeviceGetStreamPriorityRange(&leastpriority, &greatestpriority));
    CHECK_CUDA_ERROR(hipStreamCreateWithPriority(&stream1,hipStreamNonBlocking,leastpriority));
    CHECK_CUDA_ERROR(hipStreamCreateWithPriority(&stream2,hipStreamNonBlocking,greatestpriority));

    // Initialize data
    for (int i = 0; i < numOfElements; ++i) {
        h_A[i] = 1.0;
        h_B[i] = 1.0; 
    }

    CHECK_CUDA_ERROR(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream1));
    CHECK_CUDA_ERROR(hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream2));

    kernel1<<<(numOfElements+255)/256,256,0,stream1>>>(d_A,numOfElements);
    kernel2<<<(numOfElements+255)/256,256,0,stream2>>>(d_B,numOfElements);

    CHECK_CUDA_ERROR(hipStreamAddCallback(stream1,myStreamCallback,NULL,0));

    CHECK_CUDA_ERROR(hipMemcpyAsync(h_A,d_A,size,hipMemcpyDeviceToHost,stream1));
    CHECK_CUDA_ERROR(hipMemcpyAsync(h_B,d_B,size,hipMemcpyDeviceToHost,stream2));

    CHECK_CUDA_ERROR(hipStreamSynchronize(stream1));
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream2));


    // Verify result for d_A and h_A (stream1)
    for (int i = 0; i < numOfElements; ++i) {
        float expected = 2.0f;   // Initial was 1.0f, kernel adds 1.0f once
        if (fabs(h_A[i] - expected) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d in h_A! Expected %f but got %f\n", i, expected, h_A[i]);
            exit(EXIT_FAILURE);
        }
    }

    // Verify result for d_B and h_B (stream2)
    for (int i = 0; i < numOfElements; ++i) {
        float expected = 2.0f;   // Initial was 1.0f, kernel adds 1.0f once
        if (fabs(h_B[i] - expected) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d in h_B! Expected %f but got %f\n", i, expected, h_B[i]);
            exit(EXIT_FAILURE);
        }
    }

printf("Test PASSED\n");


    // Clean up
    CHECK_CUDA_ERROR(hipHostFree(h_A));
    CHECK_CUDA_ERROR(hipFree(d_A));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream1));
    CHECK_CUDA_ERROR(hipStreamDestroy(stream2));

    return 0;
}